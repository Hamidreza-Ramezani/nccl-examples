#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <stdlib.h>
#include <hip/hip_runtime_api.h>


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


int main(int argc, char* argv[])
{
  ncclComm_t comms[4];

  //managing 4 devices
  int nDev = 4;
 //int size = 32*1024*1024;
 
  int size = 1;
  int devs[4] = { 0, 1, 2, 3 };


  //allocating and initializing device buffers
  float** sendbuff = (float**)malloc(nDev * sizeof(float*));
  float** recvbuff = (float**)malloc(nDev * sizeof(float*));
  hipStream_t* s = (hipStream_t*)malloc(nDev * sizeof(hipStream_t));
  
  float* h_sendbuff = (float*)malloc(size * sizeof(float));
  memset(h_sendbuff, 127, sizeof(float) * size);
  printf("%f\n",h_sendbuff[0]);
  exit(0);
  
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc((void**)sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc((void**)recvbuff + i, size * sizeof(float)));
    CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(s+i));
    hipDeviceSynchronize();
  }

  //initializing NCCL

    //cudaMemcpy(h_sendbuff,sendbuff[0],size * sizeof(float),cudaMemcpyDefault);
    hipMemcpy(h_sendbuff,sendbuff[0],size * sizeof(float),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("%.2f",h_sendbuff[0]);
    printf("\n");
    exit(0);

   //calling NCCL communication API. Group API is required when using
   //multiple devices per thread
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i)
    NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
        comms[i], s[i]));
  NCCLCHECK(ncclGroupEnd());




  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }



  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }


  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);


  printf("Success \n");
  return 0;
}

